#include "hip/hip_runtime.h"
#include "oa.h"

/// Initialise the host and device parameters
/// Setup MTGP prng states
/// @param f Function to be evaluvated
/// @param l lower bound of the function
/// @param u upper bound of the function
OA::OA(__device__ float (*f)(const float __restrict__ &), float l, float u){
    function = f;
    bound_low = l;
    bound_high = u;
    hipMalloc((void**) &device_solution, (sizeof(float)));

    /// Allocate space for prng states on device
    hipMalloc((void **)&devMTGPStates, dimension * 
    sizeof(hiprandStateMtgp32));
     
    /// Allocate space for MTGP kernel parameters
    hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params));

    /// Reformat from predefined parameter sets to kernel format,
    /// and copy kernel parameters to device memory
    hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams);
    
    /// Initialize one state per thread block
    hiprandMakeMTGP32KernelState(devMTGPStates, 
                mtgp32dc_params_fast_11213, devKernelParams, psize, time(NULL));
}

/// The main function for WOAM
__global__ void OA::woam(){
    int myID = threadIdx.x;
    hiprandStateMtgp32 localState = devMTGPStates[myID];
    float myData[dimension],cost,costBest;
    int indexBest=myID;

    for (int j = 0; j < dimension; j++){
        /// generate data
        myData[j] = (float)(bound_low + (hiprand_uniform(&localState)* (bound_high - bound_low));
    }
    cost = function(myData);
    costBest = cost;

    getBest(&indexBest,&costBest);

    for (int k = 0; k < max_iter; k++){
        // mMSOS Component
        msos(&myData,&cost,&localState);

        costBest = cost;
        indexBest=myID;
        getBest(&indexBest,&costBest);

        // WOA Component
        woa(&myData,&cost,k,&localState,&indexBest,&costBest);

        costBest = cost;
        indexBest=myID;
        getBest(&indexBest,&costBest);
    }

}

/// Finds the best cost in the population
/// Uses a butterfly reduction
/// @param indexBest Index of the individual with minimum cost, will be updated
/// @param costBest Cost of the individual with minimum cost, will be updated
__device__ void OA::getBest(int * __restrict__ indexBest,float * __restrict__ costBest){
    float costTemp;
    int indexTemp;
    for (int i=16; i>=1; i/=2){
        costTemp = __shfl_xor_sync(0xffffffff, costBest, i, 32);
        indexTemp = __shfl_xor_sync(0xffffffff, indexBest, i,32);
        if(costTemp<costBest){
            costBest=costTemp;
            indexBest = indexTemp;
        }
    }
}

/// Get data from individual "index"
/// @param index Index of the individual to copy data from
/// @param myData Pointer for my data
/// @param myCost Pointer for my cost
/// @param data Pointer to the float array to which we will copy the other individual's data
/// @param cost Pointer to the float to which we will copy the other individual's cost
__device__ void OA::getData(const int index,const float * __restrict__ myData,const float * __restrict__ myCost,
    float * __restrict__ data,float * __restrict__ cost){
    *cost = __shfl_sync(0xffffffff, myCost,index);
    for (int i = 0; i < dimension; i++){
        data[i] = __shfl_sync(0xffffffff, myData[i],index);
    }
}

/// Update the population for msos
/// @param random_particles Array of the two random individuals picked
/// @param myData Pointer for my data
/// @param myCost Pointer for my cost
/// @param localState MTGP32 PRG state
__device__ void OA::updatePop(const int * __restrict__ random_particles,float * __restrict__ my_Data,
    float * __restrict__ my_cost, hiprandStateMtgp32 *localState){
    
    float cost_rp[2],data_rp[2*dimension];

    getData(random_particles[0],my_Data,my_cost,data_rp,&cost_rp[0]);
    getData(random_particles[1],my_Data,my_cost,&data_rp[dimension],&cost_rp[1]);

    /// Calculate Fitness
    int highFitness, lowFitness, hIndex , lIndex;

    highFitness = cost_rp[0] < cost_rp[1];
    lowFitness = !highFitness;

    /// Also the corresponding starting indices of the high and low fitness individuals
    hIndex = highFitness * dimension;
    lIndex = lowFitness * dimension;

    float my_Data_kp1[dimension];
    float my_cost_kp1;
    
    int bf1,bf2,x,y,z;
    float mv;

    bf1 = 1 + hiprand_uniform(localState) * 2;
    bf2 = 1 + hiprand_uniform(localState) * 2;

    /// Calculate the k+1 population values
    for (x = 0,y=lIndex,z=hIndex; x < dimension; x++,y++,z++){
        mv = (my_Data[x] + data_rp[z])/2;
        my_Data_kp1[x] = my_Data[x] + (hiprand_uniform(localState) * (data_rp[y] - mv*bf1));
        data_rp[x] = data_rp[z]  + (hiprand_uniform(localState) * (data_rp[y] - mv*bf2));
    }

    /// Calculate the costs
    my_cost_kp1 = function(my_Data_kp1);
    cost_rp[0] = function(data_rp);

    /// If the new cost is the minima update my individual data
    if(my_cost_kp1 < my_cost){
        my_cost = my_cost_kp1;
        for(int i = 0; i < dimension; i++)
            my_Data[i] = my_Data_kp1[i];
    }

    /// Need to implement the update of random indivdual
    int myID = threadIdx.x;
    int index=myID,indexTemp, rindex=random_particles[highFitness];
    float costTemp;
    for(int i = 0; i < 32; i++){
        costTemp = __shfl_sync(0xffffffff, cost_rp[0],i);
        indexTemp = __shfl_sync(0xffffffff, rindex,i);
        if(indexTemp == myID){
            if(my_cost<costTemp){
                index = i;
                my_cost = costTemp;
            }
        }
    }

    for (int i = 0; i < dimension; i++){
        my_Data[i] = __shfl_sync(0xffffffff, data_rp[i],index);
    }

}

/// Component Optimization Algorithm: Modified Mutualism Phase of SOS
/// @param myData Vector array of data of individual
/// @param cost Cost of myData for the given function
/// @param localState MTGP32 PRG state
__device__ void OA::msos(float * __restrict__ myData,float * __restrict__ cost,hiprandStateMtgp32 *localState){
    int random_particles[2];
    int my_index = threadIdx.x;
    random_particles[0] = int(hiprand_uniform(localState) * (psize-1))-1;
    if(random_particles[0] >= my_index)
        random_particles[0]++;
    random_particles[1] = int(hiprand_uniform(localState) * (psize-2))-1;
    if(random_particles[1] >= my_index)
        random_particles[1]++;
    if(random_particles[1] >= random_particles[0])
        random_particles[1]++;
    
    updatePop(&random_particles,myData,cost,localState);
}


/// Component Optimization Algorithm: Whale Optimization Algorithm WOA
/// @param myData Vector array of data of individual
/// @param cost Cost of myData for the given function
/// @param localState MTGP32 PRG state
__device__ void OA::woa(float * __restrict__ myData,float * __restrict__ cost,hiprandStateMtgp32 *localState,
    int current_iter,int * __restrict__ indexBest){
    
    /// Decreases linearly from 2 to 0
    float a_1 = 2.0 * (max_iter  - current_iter)/ max_iter;

    /// Decreases linearly from -1 to -2
    float a_2 = (-1.0 * (max_iter  - current_iter)/ max_iter) - 1.0;

    float l = (a_2 - 1)* hiprand_uniform(localState) + 1;
    
    float beta = hiprand_uniform(localState);
    

    /// Pick the random individual
    int index[2];
    index[0]=indexBest;
    index[1] = int(hiprand_uniform(localState) * (psize-1))-1;
    if(index[0] >= threadIdx.x)
        index[1]++;
    
    /// The arrays for data
    float * particles[2];
    float cost_p[2],data_best[dimension],data_rp[dimension];
    
    /// The pointers are assigned for the respective data
    particles[0] = data_best[0];
    particles[1] = data_rp[0];

    /// Get the data from the other threads in the warp
    getData(index[0],my_Data,my_cost,particles[0],&cost_p[0]);
    getData(index[1],my_Data,my_cost,particles[1],&cost_p[1]);

    /// The variables for values that change according to predicate
    float * d[2];
    float d_vals[3];
    d[0] = d_vals[0];
    d[1] = d_vals[2];
    float a[2],c[2],r;

    // Remeber to check the random variable distribution bounds for index
    int p,alpha;

    for (int j = 0; j < dimension; j++) {
        r = hiprand_uniform(localState);
        c[0] = 2.0 * r;
        c[1] = 1;

        d[0][0] = fabsf(c[0] * particles[0][j]-myData[j]);
        d[0][1] = fabsf(c[0] * particles[1][j]-myData[j]);
        d[1][0] = fabsf(c[1] * particles[0][j]-myData[j])

        a[0] = -(2.0 * a_1 * r - a_1);
        a[1] = powf(M_E,b * l) * cosf( 2.0 * M_PI * l);

        /// p: 0 or 1
        p = beta >= 0.5;
        /// alpha 0 when p == 0 && (abs(a) < 1)
        alpha = !p && (fabsf(a[0]) >= 1);

        myData[j] = particles[alpha][j] + a[p] * d[p][alpha];
        /// check solution bound
        if (myData[j] < bound_low){
            myData[j] = bound_low;
        } else if (myData[j] > bound_high){
            myData[j] = bound_high;
        }
    }

    cost = function(myData);
}

vector<float> OA::run(){
    vector<float> global_best_solution;

    woam<<1,psize>>(bound_low,bound_high,time(NULL));

    global_best_solution.pushback(*best_solution);
    return global_best_solution;
}