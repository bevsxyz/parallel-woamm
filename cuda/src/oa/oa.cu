#include "hip/hip_runtime.h"
#include "oa.h"

#define max_iter 30
#define psize 32
#define dimension 30
#define PI 3.14159265358979323846
#define b 0.8

/// @param f Function to be evaluvated
/// @param l lower bound of the function
/// @param u upper bound of the function
vector<float> run(int f, float l, float u){

    int blocks = 6;
    int threads = 32;
    vector<float> global_best_solution;
    hiprandStateMtgp32 *devMTGPStates;      /// State array for MTGP32 generator
    mtgp32_kernel_params *devKernelParams; /// Parameters for initialising PRG

    float host_solution[blocks],*device_solution;
    hipMalloc((void**)&device_solution, blocks* sizeof(float));

    /// Allocate space for prng states on device
    hipMalloc((void **)&devMTGPStates, blocks*threads * sizeof(hiprandStateMtgp32));
    
    /// Allocate space for MTGP kernel parameters
    hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params));

    /// Reformat from predefined parameter sets to kernel format,
    /// and copy kernel parameters to device memory
    hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams);
    
    /// Initialize one state per thread block
    hiprandMakeMTGP32KernelState(devMTGPStates, 
                mtgp32dc_params_fast_11213, devKernelParams, blocks*threads, time(NULL));

    woam<<<blocks,threads>>>(devMTGPStates,f,l,u, device_solution);

    hipMemcpy(host_solution, device_solution, blocks* (sizeof(float)), hipMemcpyDeviceToHost);
    
    float best = host_solution[0];
    for(int i = 1; i < blocks; i++){
        if(host_solution[i]<best)
            best = host_solution[i];
    }
    global_best_solution.push_back(best);
    hipFree(device_solution);
    hipFree(devMTGPStates);
    hipFree(devKernelParams);
    
    return global_best_solution;
}

/// The main function for WOAM
/// @param f Function to be evaluvated
/// @param l lower bound of the function
/// @param u upper bound of the function
__global__ void woam(hiprandStateMtgp32 *devMTGPStates,int f,float l, float u, float*solution){
    int myID = threadIdx.x;
    int bID  = blockIdx.x;
    hiprandStateMtgp32 localState = devMTGPStates[(bID*32)+myID];
    float myData[dimension],cost,costBest;
    int indexBest=myID;

    for (int j = 0; j < dimension; j++){
        /// generate data
        myData[j] = (float)(l + (hiprand_uniform(&localState)* (u - l)));
    }
    func(f,&myData[0],&cost);
    costBest = cost;

    getBest(&indexBest,&costBest);

    for (int k = 0; k < max_iter; k++){
        // mMSOS Component
        msos(f,&myData[0],&cost,&localState);

        costBest = cost;
        indexBest=myID;
        getBest(&indexBest,&costBest);

        // WOA Component
        woa(f,&myData[0],&cost,&localState,k,&indexBest, l,u);

        costBest = cost;
        indexBest=myID;
        getBest(&indexBest,&costBest);
    }
    if(myID==0)
        solution[bID] = costBest;
}

/// Finds the best cost in the population
/// Uses a butterfly reduction
/// @param indexBest Index of the individual with minimum cost, will be updated
/// @param costBest Cost of the individual with minimum cost, will be updated
__device__ void getBest(int * __restrict__ indexBest,float * __restrict__ costBest){\
    int ID = *indexBest;
    float Best = *costBest;
    float costTemp;
    int indexTemp;
    for (int i=16; i>=1; i/=2){
        costTemp = __shfl_xor_sync(0xffffffff, Best, i, 32);
        indexTemp = __shfl_xor_sync(0xffffffff, ID, i,32);
        if(costTemp<Best){
            Best = costTemp;
            ID = indexTemp;
        }
    }
    *costBest=Best;
    *indexBest = ID;
}

/// Get data from individual "index"
/// @param index Index of the individual to copy data from
/// @param myData Pointer for my data
/// @param myCost Pointer for my cost
/// @param data Pointer to the float array to which we will copy the other individual's data
/// @param cost Pointer to the float to which we will copy the other individual's cost
__device__ void getData(const int index,const float * __restrict__ myData,const float * __restrict__ myCost,
    float * __restrict__ data,float * __restrict__ cost){
    *cost = __shfl_sync(0xffffffff, *myCost,index);
    for (int i = 0; i < dimension; i++){
        data[i] = __shfl_sync(0xffffffff, myData[i],index);
    }
}

/// Update the population for msos
/// @param random_particles Array of the two random individuals picked
/// @param myData Pointer for my data
/// @param myCost Pointer for my cost
/// @param localState MTGP32 PRG state
__device__ void updatePop(int f,const int * __restrict__ random_particles,float * __restrict__ my_Data,
    float * __restrict__ my_cost, hiprandStateMtgp32 *localState){
    
    float cost_rp[2],data_rp[2*30];

    getData(random_particles[0],my_Data,my_cost,data_rp,&cost_rp[0]);
    getData(random_particles[1],my_Data,my_cost,&data_rp[dimension],&cost_rp[1]);

    /// Calculate Fitness
    int highFitness, lowFitness, hIndex , lIndex;

    highFitness = cost_rp[0] < cost_rp[1];
    lowFitness = !highFitness;

    /// Also the corresponding starting indices of the high and low fitness individuals
    hIndex = highFitness * dimension;
    lIndex = lowFitness * dimension;

    float my_Data_kp1[dimension];
    float my_cost_kp1;
    
    int bf1,bf2,x,y,z;
    float mv;

    bf1 = 1 + hiprand_uniform(localState) * 2;
    bf2 = 1 + hiprand_uniform(localState) * 2;

    /// Calculate the k+1 population values
    for (x = 0,y=lIndex,z=hIndex; x < dimension; x++,y++,z++){
        mv = __fdividef((my_Data[x] + data_rp[z]),2);
        my_Data_kp1[x] = my_Data[x] + (hiprand_uniform(localState) * (data_rp[y] - mv*bf1));
        data_rp[x] = data_rp[z]  + (hiprand_uniform(localState) * (data_rp[y] - mv*bf2));
    }

    /// Calculate the costs
    func(f,my_Data_kp1,&my_cost_kp1);
    func(f,data_rp,&cost_rp[0]);

    /// If the new cost is the minima update my individual data
    if(my_cost_kp1 < *my_cost){
        *my_cost = my_cost_kp1;
        for(int i = 0; i < dimension; i++)
            my_Data[i] = my_Data_kp1[i];
    }

    /// Need to implement the update of random indivdual
    int myID = threadIdx.x;
    int index=myID,indexTemp, rindex=random_particles[highFitness];
    float costTemp;
    for(int i = 0; i < 32; i++){
        costTemp = __shfl_sync(0xffffffff, cost_rp[0],i);
        indexTemp = __shfl_sync(0xffffffff, rindex,i);
        if(indexTemp == myID){
            if(*my_cost>costTemp){
                index = i;
                *my_cost = costTemp;
            }
        }
    }

    for (int i = 0; i < dimension; i++){
        my_Data[i] = __shfl_sync(0xffffffff, data_rp[i],index);
    }

}

/// Component Optimization Algorithm: Modified Mutualism Phase of SOS
/// @param myData Vector array of data of individual
/// @param cost Cost of myData for the given function
/// @param localState MTGP32 PRG state
__device__ void msos(int f,float * __restrict__ myData,float * __restrict__ cost,hiprandStateMtgp32 *localState){
    int random_particles[2];
    int my_index = threadIdx.x;
    random_particles[0] = int(hiprand_uniform(localState) * (psize-1))-1;
    if(random_particles[0] >= my_index)
        random_particles[0]++;
    random_particles[1] = int(hiprand_uniform(localState) * (psize-2))-1;
    if(random_particles[1] >= my_index)
        random_particles[1]++;
    if(random_particles[1] >= random_particles[0])
        random_particles[1]++;
    
    updatePop(f,&random_particles[0],myData,cost,localState);
}


/// Component Optimization Algorithm: Whale Optimization Algorithm WOA
/// @param myData Vector array of data of individual
/// @param cost Cost of myData for the given function
/// @param localState MTGP32 PRG state
__device__ void woa(int f,float * __restrict__ myData,float * __restrict__ myCost,hiprandStateMtgp32 *localState,
    int current_iter,int * __restrict__ indexBest, float bound_low, float bound_high){
    
    /// Decreases linearly from 2 to 0
    const float a_1 = 2.0 * __fdividef((max_iter  - current_iter),max_iter);

    /// Decreases linearly from -1 to -2
    const float a_2 = (-1.0 * __fdividef((max_iter  - current_iter), max_iter)) - 1.0;

    float l = (a_2 - 1)* hiprand_uniform(localState) + 1;
    
    float beta = hiprand_uniform(localState);
    

    /// Pick the random individual
    int index[2];
    index[0]=*indexBest;
    index[1] = int(hiprand_uniform(localState) * (psize-1))-1;
    if(index[0] >= threadIdx.x)
        index[1]++;

    /// The variables for values that change according to predicate
    float a[2],c[2],d,r;

    // Remeber to check the random variable distribution bounds for index
    r = hiprand_uniform(localState);
    c[0] = 2.0 * r;
    c[1] = 1;
    a[0] = -(2.0 * a_1 * r - a_1);
    a[1] = __expf(b * l) * __cosf( 2.0 * PI * l);
    
    /// p: 0 or 1
    const int p=beta >= 0.5;
    /// alpha 0 when p == 0 && (abs(a) < 1)
    const int alpha = !p && (fabsf(a[0]) >= 1);

    /// The variables for other individual which can be the best or random
    float data_p[dimension],cost_p;

    /// Get the data from the other threads in the warp

    getData(index[alpha],myData,myCost,data_p,&cost_p);

    for (int j = 0; j < dimension; j++) {
        d = fabsf(c[p] * data_p[j] - myData[j]);

        myData[j] = data_p[j] + a[p] * d;
        /// check solution bound
        if (myData[j] < bound_low){
            myData[j] = bound_low;
        } else if (myData[j] > bound_high){
            myData[j] = bound_high;
        }
    }

    func(f,myData,myCost);
}